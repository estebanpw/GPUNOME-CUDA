#include "hip/hip_runtime.h"
#include "kernels.cuh"


__global__ void kernel_index(Hash_item * hash_table, Parameters_index * params, const char * sequence) {
	// Get the index of the current element to be processed
	unsigned long global_id = blockDim.x;
	//ulong local_id = get_local_id(0);
	unsigned long kmers_in_work_item = params->kmers_per_work_item;
	unsigned long kmer_size = params->kmer_size;
	unsigned long z_value = params->z_value;
	unsigned long t_work_items = params->global_item_size;
	unsigned long offset = params->offset;
	unsigned long j, k;

	// Until reaching end of sequence
	for(j=0; j<kmers_in_work_item; j++){
		
		// Coalescent
        unsigned long pos = global_id + (j * t_work_items);
        
        
        unsigned long hash12 = 0, hash_full = 0;
		unsigned char checker = 0, multiplier = 0, val;
		
		for(k=0; k<FIXED_K; k++){
			val = (unsigned char) sequence[pos+k];
			multiplier = (val & (unsigned char) 6) >> 1;
			checker = checker | (val & (unsigned char) 8); // Verified
			hash12 += (((unsigned long) 1) << (2*k)) * (unsigned long) multiplier;
        }

        
        
        hash_full = hash12;
        
		for(k=FIXED_K; k<kmer_size; k+=z_value){
			val = (unsigned char) sequence[pos+k];
			multiplier = (val & (unsigned char) 6) >> 1;
			checker = checker | (val & (unsigned char) 8); // Verified
			hash_full += (((unsigned long) 1) << (2*k)) * (unsigned long) multiplier;
        }
        
        
		if(checker == (unsigned char) 0){ // Verified
			hash_table[0].key = hash_full;
			//hash_table[hash12].key = hash_full;
			//hash_table[hash12].pos_in_x = pos + offset;
			//atom_inc(&hash_table[hash12].repeat);
        }	
        
	}
}